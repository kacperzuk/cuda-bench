#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 1024*1024*1024/sizeof(unsigned int)  // 1GiB of uints

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void init_states(hiprandState_t *states, unsigned int seed) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  hiprand_init(seed,
              index,
              0,
              states+index);
}

__global__ void init_cache(int n, hiprandState_t *states, unsigned int* cache) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < n; i += stride) 
    cache[i] = hiprand(states+index);
}

__global__ void bench(int n, unsigned int* cache, unsigned int* r) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  unsigned int s = 0;
  for(int i = index; i < n; i += stride) 
    s += cache[i];
  *r += s;
}

int main (int argc, char *argv[]) {
  if (argc < 2) {
    printf("Provide device number!\n");
    exit(1);
  }

  int gpuNum = atoi(argv[1]);
  gpuErrchk(hipSetDevice(gpuNum));

  printf("Generating cache...\n");

  int threads = 256;
  int blocks = 1024;

  hiprandState_t* states;
  unsigned int* gpu_nums;
  hipMalloc((void**) &states, blocks*threads * sizeof(hiprandState_t));
  hipMalloc((void**) &gpu_nums, N * sizeof(unsigned int));
  init_states<<<blocks, threads>>>(states, time(0));
  init_cache<<<blocks, threads>>>(N, states, gpu_nums);

  unsigned int* res;
  unsigned int zero = 0;
  hipMalloc((void**) &res, sizeof(unsigned int));
  hipMemcpy(res, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);
  unsigned int count = 0;
  unsigned int start = time(0);
  printf("Generated cache.\n");
  while(true) {
    bench<<<blocks, threads>>>(N, gpu_nums, res);
    count++;
    if(time(0) - start > 5) {
      printf(" m - cu%d %d\n", gpuNum, count);
      count = 0;
      start = time(0);
    }
  }
  hipFree(states);
  hipFree(gpu_nums);
  hipFree(res);

  return 0;
}
